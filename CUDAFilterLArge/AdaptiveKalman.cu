#include "hip/hip_runtime.h"
#include <chrono>
#include <iostream>
#include "hipblas.h"
#include "C:\Users\62793\CSC591\DirectedStudy\data\largeMeasurements.hpp"

#define NUM_OF_MEASUREMENTS 500
#define WINDOW_SIZE 40
#define THREADS_FOR_RED 32
#define STATE_TRANS 0
#define CONTROL_MATRIX 4096
#define IDENTITY_MATRIX 8192
#define MEASUREMENT_NOISE 12288
#define STATE_TRANS_TRANSPOSE 16384
#define IDENTITY_MATRIX_TRANSPOSE 20480 //I realize this is just the identity again, wanted to include it for size anyway

void printMatrix(float *a, int rows, int cols){
    for(int i = 0; i < rows; i++){
        for(int j = 0; j <cols;j++){
            if(i < 4 && j <4){
                printf("%6.3f  ",a[j * cols + i]);
            }
        }
        if(i < 4 ){
            std::cout << std::endl;
        }
    }
    std::cout << std::endl;
}

void checkCudaErrors(){
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }
}

void checkCublasError(hipblasStatus_t stat,int line){
    if(stat != HIPBLAS_STATUS_SUCCESS){
        printf("CUBLAS error: %d on %d\n",stat,line);
        exit(-1);
    }
}

__global__ void elementSubtractBMinusA(float *a, float *b, int rows, int cols){
    int const index = threadIdx.x + blockIdx.x * blockDim.x;
    //printf("%d\n",index);
    if(index < rows*cols){
        
        a[index] = b[index] - a[index];
        //printf("%d : %f\n",index,a[index]);
    }
}

__global__ void calcMean(float *a,float *b,int rows,int cols,int stackHeight){
    __shared__ float redData[THREADS_FOR_RED];
    
    if(threadIdx.x < stackHeight/2){
        redData[threadIdx.x] = a[((threadIdx.x*2 ) * (rows*cols)) +blockIdx.x] + a[((threadIdx.x*2 + 1)*(rows*cols))+blockIdx.x];
    }else{
        redData[threadIdx.x] = 0;
    }

    __syncthreads();

    for(int i = 1; i < blockDim.x; i *= 2){
        if( threadIdx.x  % (2 *i) == 0){
            redData[threadIdx.x] += redData[threadIdx.x + i];
        }
        __syncthreads();
    }
    b[blockIdx.x] = redData[0]/stackHeight;
}

__global__ void marker(int *i){
    int const index = threadIdx.x + blockIdx.x * blockDim.x;
    i += index;
}

void checkMatrix(float *r, float *result, int size, int rows, int cols){
    hipDeviceSynchronize();
    hipMemcpy(result,r,size,hipMemcpyDeviceToHost);
    printMatrix(result,rows,cols);
    checkCudaErrors();
}

float kalman(float measurements[][4096],int num_measurements, int measurement_rows, int measurement_columns){
    float *dev_measurement,*dev_result, *dev_process_noise,*dev_invert_result,*dev_process_error,*dev_kalman_gain,*dev_temp;
    float *result;
    int *dev_info;
    float *dev_residual;
    float *dev_innovation_bank;
    float *dev_batch_consts;
    float *dev_kalman_gain_final;
    float *dev_temp2;
    float *dev_running_sum;
    int *dev_pivot;

    int four_by_four_float_array_size = measurement_columns * measurement_rows* sizeof(float);

    //result = (float*) malloc(four_by_four_float_array_size);
    
    hipMalloc((void **) &dev_batch_consts,four_by_four_float_array_size*6);
    hipMalloc((void **) &dev_measurement,four_by_four_float_array_size);
    hipMalloc((void **) &dev_result,four_by_four_float_array_size);
    hipMalloc((void **) &dev_process_noise,four_by_four_float_array_size);
    hipMalloc((void **) &dev_invert_result,four_by_four_float_array_size);
    hipMalloc((void **) &dev_process_error,four_by_four_float_array_size);
    hipMalloc((void **) &dev_kalman_gain,four_by_four_float_array_size);
    hipMalloc((void **) &dev_temp,four_by_four_float_array_size);
    hipMalloc((void **) &dev_info,sizeof(int));
    hipMalloc((void **) &dev_residual,four_by_four_float_array_size);
    hipMalloc((void **) &dev_innovation_bank,four_by_four_float_array_size*WINDOW_SIZE);
    hipMalloc((void **) &dev_temp2,four_by_four_float_array_size);
    hipMalloc((void **) &dev_kalman_gain_final,four_by_four_float_array_size);
    hipMalloc((void **) &dev_running_sum,four_by_four_float_array_size);
    hipMalloc((void **) &dev_pivot,measurement_rows*sizeof(int));
    hipHostMalloc((void **) &result, four_by_four_float_array_size);

    float *A[] = { dev_temp };
    float** A_d;
    hipMalloc<float*>(&A_d,sizeof(A));
    hipMemcpy(A_d,A,sizeof(A),hipMemcpyHostToDevice);
    checkCudaErrors();

    float *C[] = { dev_invert_result };
    float** C_d;
    hipMalloc<float*>(&C_d,sizeof(C));
    hipMemcpy(C_d,C,sizeof(C),hipMemcpyHostToDevice);
    checkCudaErrors();
   
    checkCudaErrors();

    hipMemcpyAsync(dev_batch_consts,batched_const_matrices,four_by_four_float_array_size*6,hipMemcpyHostToDevice);
    hipMemcpyAsync(dev_running_sum,mean_init,four_by_four_float_array_size,hipMemcpyHostToDevice);
    hipMemcpyAsync(dev_result,measurements[0],four_by_four_float_array_size,hipMemcpyHostToDevice);
    hipMemcpy(dev_process_error,&dev_batch_consts[IDENTITY_MATRIX],four_by_four_float_array_size,hipMemcpyDeviceToDevice);
    hipMemcpyAsync(dev_process_noise,&dev_batch_consts[IDENTITY_MATRIX],four_by_four_float_array_size,hipMemcpyDeviceToDevice);

    checkCudaErrors();

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    const float alpha = 1;
    const float beta = 0;

    hipblasStatus_t stat;

    hipStream_t stream1;
    hipStream_t stream2;
    hipStream_t stream3;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);


    auto start_time = std::chrono::system_clock::now();
    for(int i=1;i<num_measurements;i++){
        hipMemcpyAsync(dev_measurement,measurements[i],four_by_four_float_array_size,hipMemcpyHostToDevice,stream1);
        checkCudaErrors(); 

        //predict
        //A*(x-1)
        hipblasSetStream(handle,stream2);
        stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, measurement_rows, measurement_columns, measurement_columns, &alpha, &dev_batch_consts[STATE_TRANS], measurement_rows, dev_result, measurement_rows, &beta, dev_result, measurement_rows);
        checkCublasError(stat,1);
        //+Buk
        stat = hipblasSaxpy(handle, measurement_rows*measurement_columns,&alpha,&dev_batch_consts[CONTROL_MATRIX],1,dev_result,1);
        checkCublasError(stat,2);
        
        hipblasSetStream(handle,stream3);
        //A*(p-1)
        stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, measurement_rows, measurement_columns, measurement_columns, &alpha,  &dev_batch_consts[STATE_TRANS], measurement_rows, dev_process_error, measurement_rows, &beta, dev_process_error, measurement_rows);
        checkCublasError(stat,3);
        //*At
        stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, measurement_rows, measurement_columns, measurement_columns, &alpha, dev_process_error, measurement_rows,  &dev_batch_consts[STATE_TRANS_TRANSPOSE], measurement_rows, &beta, dev_process_error, measurement_rows);
        checkCublasError(stat,4);
        //+Q
        stat = hipblasSaxpy(handle, measurement_rows*measurement_columns,&alpha,dev_process_noise,1,dev_process_error,1);
        checkCublasError(stat,5);
        
        hipblasSetStream(handle,stream2);
        //Calculate Residual
        //H*Xp
        stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, measurement_rows, measurement_columns, measurement_columns, &alpha,  &dev_batch_consts[IDENTITY_MATRIX], measurement_rows, dev_result, measurement_rows, &beta, dev_residual, measurement_rows);
        checkCublasError(stat,12);

        hipStreamSynchronize(stream1);
        //Y-
        elementSubtractBMinusA<<<measurement_rows,measurement_rows,0,stream2>>>(dev_residual,dev_measurement,measurement_rows,measurement_columns);
        checkCudaErrors();

        //Adaptation
        //Res*Tranpose(res)
        hipblasSetStream(handle,stream2);
        stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, measurement_rows, measurement_columns, measurement_columns, &alpha, dev_residual, measurement_rows, dev_residual, measurement_rows, &beta, dev_temp2, measurement_rows);
        checkCublasError(stat,17);
        //Move Residual into bank
        int offset = ((i-1)%WINDOW_SIZE) * measurement_rows* measurement_columns;
        hipMemcpyAsync(dev_innovation_bank + offset,dev_temp2,four_by_four_float_array_size,hipMemcpyDeviceToDevice,stream2);
        checkCudaErrors();
        
        if(i>=WINDOW_SIZE){
            calcMean<<<measurement_rows*measurement_columns,THREADS_FOR_RED,(THREADS_FOR_RED)*sizeof(float),stream2>>>(dev_innovation_bank,dev_temp2,measurement_rows,measurement_columns,WINDOW_SIZE);
            checkCudaErrors();  

            hipblasSetStream(handle,stream2);
            stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, measurement_rows, measurement_columns, measurement_columns, &alpha, dev_kalman_gain_final, measurement_rows, dev_temp2, measurement_rows, &beta, dev_temp2, measurement_rows);
            checkCublasError(stat,18);

            stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, measurement_rows, measurement_columns, measurement_columns, &alpha, dev_temp2, measurement_rows, dev_kalman_gain_final, measurement_rows, &beta, dev_process_noise, measurement_rows);
            checkCublasError(stat,19);
        }


        //update
        //P*Ht
        hipblasSetStream(handle,stream3);
        stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, measurement_rows, measurement_columns, measurement_columns, &alpha, dev_process_error, measurement_rows,  &dev_batch_consts[IDENTITY_MATRIX_TRANSPOSE], measurement_rows, &beta, dev_kalman_gain, measurement_rows);
        checkCublasError(stat,6);

        hipStreamSynchronize(stream3);
        hipblasSetStream(handle,stream1);
        //H*P*Ht
        stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, measurement_rows, measurement_columns, measurement_columns, &alpha,  &dev_batch_consts[IDENTITY_MATRIX], measurement_rows, dev_kalman_gain, measurement_rows, &beta, dev_temp, measurement_rows);
        checkCublasError(stat,7);
        //+R
        stat = hipblasSaxpy(handle, measurement_rows*measurement_columns,&alpha, &dev_batch_consts[MEASUREMENT_NOISE],1,dev_temp,1);
        checkCublasError(stat,__LINE__);

        stat = hipblasSgetrfBatched(handle,measurement_rows,A_d,measurement_rows,dev_pivot,dev_info,1);
        checkCublasError(stat,__LINE__);

        stat = hipblasSgetriBatched(handle,measurement_rows,A_d,measurement_rows,dev_pivot,C_d,measurement_rows,dev_info,1);
        //K = (P*Ht)/(H*P*Ht+R)
        checkCublasError(stat,__LINE__);
        hipDeviceSynchronize();

        stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, measurement_rows, measurement_columns, measurement_columns, &alpha, dev_kalman_gain, measurement_rows, dev_invert_result, measurement_rows, &beta, dev_kalman_gain_final, measurement_rows);
        checkCublasError(stat,11);

        
        hipStreamSynchronize(stream1);
        //K*Residual
        stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, measurement_rows, measurement_columns, measurement_columns, &alpha, dev_kalman_gain_final, measurement_rows, dev_residual, measurement_rows, &beta, dev_temp, measurement_rows);
        checkCublasError(stat,13);
        //+Xp
        stat = hipblasSaxpy(handle, measurement_rows*measurement_columns,&alpha,dev_temp,1,dev_result,1);
        checkCublasError(stat,14);

        hipblasSetStream(handle,stream2);
        //K*H
        stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, measurement_rows, measurement_columns, measurement_columns, &alpha, dev_kalman_gain_final, measurement_rows,  &dev_batch_consts[IDENTITY_MATRIX], measurement_rows, &beta, dev_temp2, measurement_rows);
        checkCublasError(stat,15);
        //I-
        elementSubtractBMinusA<<<measurement_rows,measurement_rows,0,stream2>>>(dev_temp2, &dev_batch_consts[IDENTITY_MATRIX],measurement_rows,measurement_columns);
        checkCudaErrors();
        //*P
        stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, measurement_rows, measurement_columns, measurement_columns, &alpha, dev_temp2, measurement_rows, dev_process_error, measurement_rows, &beta, dev_process_error, measurement_rows);
        checkCublasError(stat,16);

        //hipDeviceSynchronize();
        hipMemcpyAsync(result,dev_result,four_by_four_float_array_size,hipMemcpyDeviceToHost,stream1);
        checkCudaErrors();
        //printMatrix(result,measurement_rows,measurement_columns);  
    }
    auto end_time = std::chrono::system_clock::now();
    auto elapsed_time = std::chrono::duration_cast< std::chrono::milliseconds >( end_time - start_time ).count()/static_cast<float>(100);
    //std::cout << "average time per measurment: " << elapsed_time<< " ms" << std::endl;

    hipFree(dev_measurement);
    hipFree(dev_result);
    hipFree(dev_process_noise);
    hipFree(dev_invert_result);
    hipFree(dev_process_error);
    hipFree(dev_kalman_gain);
    hipFree(dev_temp);
    hipFree(dev_info);
    hipFree(dev_innovation_bank);
    hipFree(dev_residual);
    hipHostFree(result);
    hipblasDestroy(handle);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipStreamDestroy(stream3);
    checkCudaErrors();
    return elapsed_time;
}

int main(){
    float tpm = 0;
    auto start_time = std::chrono::system_clock::now();
    for(int i = 0; i < 100; i++){
        tpm += kalman(measurements,NUM_OF_MEASUREMENTS,64,64);
    }
    auto end_time = std::chrono::system_clock::now();
    auto elapsed_time = std::chrono::duration_cast< std::chrono::microseconds >( end_time - start_time );
    std::cout << "average time per run: " << elapsed_time.count() / static_cast< float >( NUM_OF_MEASUREMENTS)<< " us" << std::endl;
    std::cout << "average time per measurment: " << tpm/ static_cast< float >( NUM_OF_MEASUREMENTS)<< " ms" << std::endl;

}

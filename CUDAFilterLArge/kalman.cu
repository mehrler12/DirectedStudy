#include "hip/hip_runtime.h"
#include <chrono>
#include <iostream>
#include "hipblas.h"
#include "C:\Users\62793\CSC591\DirectedStudy\data\largeMeasurements.hpp"

#define NUM_OF_MEASUREMENTS 100
#define NUM_OF_MEASURES_IN_FILE 2
#define WINDOW_SIZE 20
#define MEASUREMENT_SIZE 1024
#define STATE_TRANS 0
#define CONTROL_MATRIX MEASUREMENT_SIZE*MEASUREMENT_SIZE
#define IDENTITY_MATRIX MEASUREMENT_SIZE*MEASUREMENT_SIZE *2
#define MEASUREMENT_NOISE MEASUREMENT_SIZE*MEASUREMENT_SIZE *3
#define STATE_TRANS_TRANSPOSE MEASUREMENT_SIZE*MEASUREMENT_SIZE *4
#define IDENTITY_MATRIX_TRANSPOSE MEASUREMENT_SIZE*MEASUREMENT_SIZE *5 //I realize this is just the identity again, wanted to include it for size anyway

void printMatrix(float *a, int rows, int cols){
    for(int i = 0; i < rows; i++){
        for(int j = 0; j <cols;j++){
            if(i < 4 && j <4){
                printf("%6.3f  ",a[j * cols + i]);
            }
        }
        if(i < 4 ){
            std::cout << std::endl;
        }
    }
    std::cout << std::endl;
}

void checkCudaErrors(int line){
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        printf("CUDA error: %s on line %d\n", hipGetErrorString(error),line);
        exit(-1);
    }
}

void checkCublasError(hipblasStatus_t stat,int line){
    if(stat != HIPBLAS_STATUS_SUCCESS){
        printf("CUBLAS error: %d on %d\n",stat,line);
        exit(-1);
    }
}

__global__ void elementSubtractBMinusA(float *a, float *b, int rows, int cols){
    int const index = threadIdx.x + blockIdx.x * blockDim.x;
    //printf("%d\n",index);
    if(index < rows*cols){
        
        a[index] = b[index] - a[index];
        //printf("%d : %f\n",index,a[index]);
    }
}

// __global__ void calcMean(float *a,float *b,int rows,int cols,int stackHeight){
//     __shared__ float redData[THREADS_FOR_RED];
    
//     if(threadIdx.x < stackHeight/2){
//         redData[threadIdx.x] = a[((threadIdx.x*2 ) * (rows*cols)) +blockIdx.x] + a[((threadIdx.x*2 + 1)*(rows*cols))+blockIdx.x];
//     }else{
//         redData[threadIdx.x] = 0;
//     }

//     __syncthreads();

//     for(int i = 1; i < blockDim.x; i *= 2){
//         if( threadIdx.x  % (2 *i) == 0){
//             redData[threadIdx.x] += redData[threadIdx.x + i];
//         }
//         __syncthreads();
//     }
//     b[blockIdx.x] = redData[0]/stackHeight;
// }


__global__ void marker(int *i){
    int const index = threadIdx.x + blockIdx.x * blockDim.x;
    i += index;
}

void checkMatrix(float *r, float *result, int size, int rows, int cols){
    hipDeviceSynchronize();
    hipMemcpy(result,r,size,hipMemcpyDeviceToHost);
    printMatrix(result,rows,cols);
    checkCudaErrors(__LINE__);
}

float kalman(float measurements[][MEASUREMENT_SIZE*MEASUREMENT_SIZE],int num_measurements, int measurement_rows, int measurement_columns){
    float *dev_measurement,*dev_result, *dev_process_noise,*dev_invert_result,*dev_process_error,*dev_kalman_gain,*dev_temp;
    float *result;
    int *dev_info;
    float *dev_residual;
    float *dev_batch_consts;
    float *dev_temp2;
    int *dev_pivot;

    int four_by_four_float_array_size = measurement_columns * measurement_rows* sizeof(float);

    //result = (float*) malloc(four_by_four_float_array_size);
    
    hipMalloc((void **) &dev_batch_consts,four_by_four_float_array_size*6);
    hipMalloc((void **) &dev_measurement,four_by_four_float_array_size);
    hipMalloc((void **) &dev_result,four_by_four_float_array_size);
    hipMalloc((void **) &dev_process_noise,four_by_four_float_array_size);
    hipMalloc((void **) &dev_invert_result,four_by_four_float_array_size);
    hipMalloc((void **) &dev_process_error,four_by_four_float_array_size);
    hipMalloc((void **) &dev_kalman_gain,four_by_four_float_array_size);
    hipMalloc((void **) &dev_temp,four_by_four_float_array_size);
    hipMalloc((void **) &dev_info,sizeof(int));
    hipMalloc((void **) &dev_residual,four_by_four_float_array_size);
    hipMalloc((void **) &dev_temp2,four_by_four_float_array_size);
    hipMalloc((void **) &dev_pivot,measurement_rows*sizeof(int));
    hipHostMalloc((void **) &result, four_by_four_float_array_size);
    float *A[] = { dev_temp };
    float** A_d;
    hipMalloc<float*>(&A_d,sizeof(A));
    hipMemcpy(A_d,A,sizeof(A),hipMemcpyHostToDevice);
    checkCudaErrors(__LINE__);

    float *C[] = { dev_invert_result };
    float** C_d;
    hipMalloc<float*>(&C_d,sizeof(C));
    hipMemcpy(C_d,C,sizeof(C),hipMemcpyHostToDevice);
    checkCudaErrors(__LINE__);
   
    checkCudaErrors(__LINE__);

    hipMemcpyAsync(dev_batch_consts,batched_const_matrices,four_by_four_float_array_size*6,hipMemcpyHostToDevice);
    hipMemcpyAsync(dev_result,measurements[0],four_by_four_float_array_size,hipMemcpyHostToDevice);
    hipMemcpy(dev_process_error,&dev_batch_consts[IDENTITY_MATRIX],four_by_four_float_array_size,hipMemcpyDeviceToDevice);
    hipMemcpyAsync(dev_process_noise,&dev_batch_consts[IDENTITY_MATRIX],four_by_four_float_array_size,hipMemcpyDeviceToDevice);

    checkCudaErrors(__LINE__);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    const float alpha = 1;
    const float beta = 0;

    hipblasStatus_t stat;

    hipStream_t stream1;
    hipStream_t stream2;
    hipStream_t stream3;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);


    auto start_time = std::chrono::system_clock::now();
    for(int j=1;j<num_measurements;j++){
        int i = j % NUM_OF_MEASURES_IN_FILE;
        //printf("Iteration %d\n",i);
        hipMemcpyAsync(dev_measurement,measurements[i],four_by_four_float_array_size,hipMemcpyHostToDevice,stream1);
        checkCudaErrors(__LINE__); 

        //predict
        //A*(x-1)
        hipblasSetStream(handle,stream2);
        stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, measurement_rows, measurement_columns, measurement_columns, &alpha, &dev_batch_consts[STATE_TRANS], measurement_rows, dev_result, measurement_rows, &beta, dev_result, measurement_rows);
        checkCublasError(stat,1);
        //+Buk
        stat = hipblasSaxpy(handle, measurement_rows*measurement_columns,&alpha,&dev_batch_consts[CONTROL_MATRIX],1,dev_result,1);
        checkCublasError(stat,2);
        
        hipblasSetStream(handle,stream3);
        //A*(p-1)
        stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, measurement_rows, measurement_columns, measurement_columns, &alpha,  &dev_batch_consts[STATE_TRANS], measurement_rows, dev_process_error, measurement_rows, &beta, dev_process_error, measurement_rows);
        checkCublasError(stat,3);
        //*At
        stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, measurement_rows, measurement_columns, measurement_columns, &alpha, dev_process_error, measurement_rows,  &dev_batch_consts[STATE_TRANS_TRANSPOSE], measurement_rows, &beta, dev_process_error, measurement_rows);
        checkCublasError(stat,4);
        //+Q
        stat = hipblasSaxpy(handle, measurement_rows*measurement_columns,&alpha,dev_process_noise,1,dev_process_error,1);
        checkCublasError(stat,5);
        
        hipblasSetStream(handle,stream2);
        //Calculate Residual
        //H*Xp
        stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, measurement_rows, measurement_columns, measurement_columns, &alpha,  &dev_batch_consts[IDENTITY_MATRIX], measurement_rows, dev_result, measurement_rows, &beta, dev_residual, measurement_rows);
        checkCublasError(stat,12);

        hipStreamSynchronize(stream1);
        //Y-
        elementSubtractBMinusA<<<measurement_rows,measurement_rows,0,stream2>>>(dev_residual,dev_measurement,measurement_rows,measurement_columns);
        checkCudaErrors(__LINE__);


        //update
        //P*Ht
        hipblasSetStream(handle,stream3);
        stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, measurement_rows, measurement_columns, measurement_columns, &alpha, dev_process_error, measurement_rows,  &dev_batch_consts[IDENTITY_MATRIX_TRANSPOSE], measurement_rows, &beta, dev_kalman_gain, measurement_rows);
        checkCublasError(stat,6);

        hipStreamSynchronize(stream3);
        hipblasSetStream(handle,stream1);
        //H*P*Ht
        stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, measurement_rows, measurement_columns, measurement_columns, &alpha,  &dev_batch_consts[IDENTITY_MATRIX], measurement_rows, dev_kalman_gain, measurement_rows, &beta, dev_temp, measurement_rows);
        checkCublasError(stat,7);
        //+R
        stat = hipblasSaxpy(handle, measurement_rows*measurement_columns,&alpha, &dev_batch_consts[MEASUREMENT_NOISE],1,dev_temp,1);
        checkCublasError(stat,__LINE__);

        stat = hipblasSgetrfBatched(handle,measurement_rows,A_d,measurement_rows,dev_pivot,dev_info,1);
        checkCublasError(stat,__LINE__);

        stat = hipblasSgetriBatched(handle,measurement_rows,A_d,measurement_rows,dev_pivot,C_d,measurement_rows,dev_info,1);
        //K = (P*Ht)/(H*P*Ht+R)
        checkCublasError(stat,__LINE__);
        hipDeviceSynchronize();

        stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, measurement_rows, measurement_columns, measurement_columns, &alpha, dev_kalman_gain, measurement_rows, dev_invert_result, measurement_rows, &beta, dev_kalman_gain, measurement_rows);
        checkCublasError(stat,11);

        
        hipStreamSynchronize(stream1);
        //K*Residual
        stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, measurement_rows, measurement_columns, measurement_columns, &alpha, dev_kalman_gain, measurement_rows, dev_residual, measurement_rows, &beta, dev_temp, measurement_rows);
        checkCublasError(stat,13);
        //+Xp
        stat = hipblasSaxpy(handle, measurement_rows*measurement_columns,&alpha,dev_temp,1,dev_result,1);
        checkCublasError(stat,14);

        hipblasSetStream(handle,stream2);
        //K*H
        stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, measurement_rows, measurement_columns, measurement_columns, &alpha, dev_kalman_gain, measurement_rows,  &dev_batch_consts[IDENTITY_MATRIX], measurement_rows, &beta, dev_temp2, measurement_rows);
        checkCublasError(stat,15);
        //I-
        elementSubtractBMinusA<<<measurement_rows,measurement_rows,0,stream2>>>(dev_temp2, &dev_batch_consts[IDENTITY_MATRIX],measurement_rows,measurement_columns);
        checkCudaErrors(__LINE__);
        //*P
        stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, measurement_rows, measurement_columns, measurement_columns, &alpha, dev_temp2, measurement_rows, dev_process_error, measurement_rows, &beta, dev_process_error, measurement_rows);
        checkCublasError(stat,16);

        //hipDeviceSynchronize();
        hipMemcpyAsync(result,dev_result,four_by_four_float_array_size,hipMemcpyDeviceToHost,stream1);
        checkCudaErrors(__LINE__);
        //printMatrix(result,measurement_rows,measurement_columns);  
    }
    auto end_time = std::chrono::system_clock::now();
    auto elapsed_time = std::chrono::duration_cast< std::chrono::milliseconds >( end_time - start_time ).count()/static_cast<float>(NUM_OF_MEASUREMENTS);
    //std::cout << "average time per measurment: " << elapsed_time<< " ms" << std::endl;

    hipFree(dev_measurement);
    hipFree(dev_result);
    hipFree(dev_process_noise);
    hipFree(dev_invert_result);
    hipFree(dev_process_error);
    hipFree(dev_kalman_gain);
    hipFree(dev_temp);
    hipFree(dev_temp2);
    hipFree(dev_info);
    hipFree(dev_residual);
    hipFree(dev_batch_consts);
    hipFree(dev_pivot);
    hipHostFree(result);
    hipblasDestroy(handle);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipStreamDestroy(stream3);
    checkCudaErrors(__LINE__);
    return elapsed_time;
}

int main(){
    float tpm = 0;
    auto start_time = std::chrono::system_clock::now();
    for(int i = 0; i < 1000; i++){
        tpm += kalman(measurements,NUM_OF_MEASUREMENTS,MEASUREMENT_SIZE,MEASUREMENT_SIZE);
    }
    auto end_time = std::chrono::system_clock::now();
    auto elapsed_time = std::chrono::duration_cast< std::chrono::milliseconds >( end_time - start_time );
    std::cout << "average time per run: " << elapsed_time.count() / static_cast< float >( 1000)<< " ms" << std::endl;
    std::cout << "average time per measurment: " << tpm/ static_cast< float >( 1000)<< " ms" << std::endl;

}
